#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void print_all_idx()
{
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int tidz = threadIdx.z;

    int bidx = blockIdx.x;
    int bidy = blockIdx.y;
    int bidz = blockIdx.z;

    int gidx = gridDim.x;
    int gidy = gridDim.y;
    int gidz = gridDim.z;

    printf("[DEVICE] threadIdx.x: %d, blockIdx.x: %d, gridDim.x: %d \n", tidx, bidx, gidx);
    printf("[DEVICE] threadIdx.y: %d, blockIdx.y: %d, gridDim.y: %d \n", tidy, bidy, gidx);
    printf("[DEVICE] threadIdx.z: %d, blockIdx.z: %d, gridDim.z: %d \n", tidz, bidz, gidz);
}

int main()
{
    dim3 blockSize(4, 4, 4);
    dim3 gridSize(2, 2, 2);

    //Punteros (Saber cuales usar en GPU y en CPU)
    // Initialization
    int* c_host; //Host o CPU
    int* a_host;
    int* b_host;

    int* c_device; //Device o GPU
    int* a_device;
    int* b_device;

    const int data_count = 10000;
    const int data_size = data_count * sizeof(int);

    c_host = (int*)malloc(data_size);
    a_host = (int*)malloc(data_size);
    b_host = (int*)malloc(data_size);

    //Almacenar el la memoria nuestras variables
    //Ayuda a reservar memoria en la memoria de video
    // Memory allocation
    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);
    
    // Transfer CPU host to GPU device
    hipMemcpy(c_device, c_host, data_size, hipMemcpyHostToDevice);
    hipMemcpy(a_device, a_host, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, data_size, hipMemcpyHostToDevice);

    // Launch to kernel
    print_all_idx << <gridSize , blockSize >> > (); 

    // Transfer GPU device to CPU host
    hipMemcpy(c_device, c_host, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_device, a_host, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_device, b_host, data_size, hipMemcpyDeviceToHost);

    //Limpieza de la memoria
    hipDeviceReset();
    hipFree(c_device);
    hipFree(a_device);
    hipFree(b_device);

    return 0;
}

/*int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/
